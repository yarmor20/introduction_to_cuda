﻿
#include "hip/hip_runtime.h"

#include <stdio.h>


__global__ void add(int *c, int *a, int *b) {
    *c = *a + *b;
}

int main() {
    int a = 5, b = 4, c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c

    // Allocate space for device copies of a, b, c
    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_c, sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<1, 1>>>(d_c, d_a, d_b);

    // Copy result back to host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    printf("Result: %d\n", c);
    return 0;
}

